#include "hip/hip_runtime.h"
#include <iostream>  
#include <math.h>
#include <arrayfire.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <fstream>
#include <algorithm> 
#include <chrono> 
#include <unistd.h>
#include <limits>
#include <set>
using namespace af;
using namespace std;

const int N = 200;  //Number of Data Samples
const string dir = "/scratch/mw4355/Radiation/";  //Path of the Dataset
const int dim =600*248*248;  //Dimension of Each Data Sample
double time_io = 0;

//+++++++++++++++++++++++++ Reading Functions Area +++++++++++++++++++++++++++++
array readData_Isabel(int file_num){
    string num;
    FILE * pFile;
    if (file_num < 10){
        num = "0"+to_string(file_num);
    }else{
        num=to_string(file_num);
    }
    string path = dir+"TCf"+num+"converted.bin";
    float* data = new float[dim];

    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io += TimerGetRTime(0);

    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        if(data[i] > 100 or data[i] < -100){
            data_[i] = 100;
        }
        else{
            data_[i] = data[i];
        }
    }
    fclose(pFile);
    array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

array readData_Vortex(int file_num){
    string num;
    FILE * pFile;
    if(file_num == 65){
        num = to_string(64);
    }else if(file_num == 24){
        num = to_string(25);
    }else{
        num = to_string(file_num);
    }
    string path = dir+"vorts"+num+"converted.data";
    float* data = new float[dim];

    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);

    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        data_[i] = data[i];
    }
    fclose(pFile);
    array oneStep(1, dim, data_);

    delete[] data;
    delete[] data_;
    return oneStep;
}

array readData_Radiation(int file_num){
    string num;
    FILE * pFile;

    file_num = file_num-1;
    if (file_num < 10){
        num = "000" + to_string(file_num);
    }else if(file_num < 100){
        num = "00" + to_string(file_num);
    }else{
        num= "0" + to_string(file_num);
    }
    string path = dir+"temperature"+num+".raw";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        if(data[i] > 23000 or data[i] < 70){
            data_[i] = 0;
        }else{
            data_[i] = data[i];
        }
    }
    fclose(pFile);
    array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

array readData_TeraShake(int file_num){
    FILE* pFile;
    string path = dir + "TeraShake" + to_string(file_num) + ".bin";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        data_[i] = data[i];
    }
    fclose(pFile);
    array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

//+++++++++++++++++++++++++ Algorithm ++++++++++++++++++++++++++++++++++++++++
double loss(array& ATB, array& ATA, array& bT, 
    array& a, double& tr_BTB, bool two_near){
    int d = bT.dims(1);
    double temp;
    double* inv_CPU_1 = new double[2];
    double* inv_CPU_2 = new double[2];
    double* ATB_CPU_1 = new double[d];
    double* ATB_CPU_2 = new double[d];
    temp = norm(bT);
    tr_BTB += temp*temp;
    ATB += matmul(a, bT);
    ATA += matmul(a, transpose(a));
    if (two_near){return 0;}
    array ATA_inv = inverse(ATA);
    ATB(0, span).host(ATB_CPU_1);
    ATB(1, span).host(ATB_CPU_2);
    ATA_inv(0, span).host(inv_CPU_1);
    ATA_inv(1, span).host(inv_CPU_2);
    double tr;
    for(int i = 0; i < d; i++){
        if(i == 0){
            tr = ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
            tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
            continue;
        }
        tr += ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
        tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
    }
    delete [] ATB_CPU_1;
    delete [] ATB_CPU_2;
    delete [] inv_CPU_1;
    delete [] inv_CPU_2;
    return tr_BTB - tr;
}


void valid_range(double & upper, double & lower){

    array a, bT, ATA_1, ATB_1, ATA, ATB;
    double cur_loss1 = numeric_limits<double>::infinity();
    int seg1 = 0;
    double tr_BTB[2];

    a = constant(1,2,1,f64);
    bT = readData_TeraShake(1);
    ATA = matmul(a,transpose(a));
    ATB = matmul(a,bT);
    tr_BTB[1] = norm(bT);
    tr_BTB[1] *= tr_BTB[1];

    ATA_1 = matmul(a,transpose(a));
    ATB_1 = matmul(a,bT);
    tr_BTB[0] = norm(bT);
    tr_BTB[0] *= tr_BTB[0];
    seg1 = 1;

    double t = 0.0;
    double s = 0.0;
    bool is_set = false;

    for(int i = 1; i < N+1; ++i){

        a(0,0) = i;
        bT = readData_TeraShake(i);
        s = norm(bT);
        if(s != 0 && is_set == false){
            is_set = true;
            t = s * s * 10000000000000000;
            cout << "Threshold: " << t << endl;
        }

        if(i == 1){
            continue;
        }

        if(seg1 < 3){
            cur_loss1 = loss(ATB_1, ATA_1, bT, a, tr_BTB[0], true);
            seg1 += 1;
        }else{
            cur_loss1 = loss(ATB_1, ATA_1, bT, a, tr_BTB[0], false);
            seg1 += 1;
        }

        if(i < 3){
            upper=loss(ATB, ATA, bT, a, tr_BTB[1], true);
        }else{
            upper=loss(ATB, ATA, bT, a, tr_BTB[1], false);
        }

        if (t == 0){
            continue;
        }

        if(cur_loss1 > t){

            lower = min(lower, cur_loss1);
            ATA_1 = matmul(a,transpose(a));
            ATB_1 = matmul(a,bT);
            tr_BTB[0] = norm(bT);
            tr_BTB[0] *= tr_BTB[0];
            seg1 = 1;
        }
    }

}


double greedy_construct(double alpha, int& num_segment){

    array a, bT, ATA, ATB;
    int num_points = 0;
    double acc_loss = 0;
    double cur_loss = 0;
    double prev_loss = 0;
    double tr_BTB = 0;
    vector<int> solu;

    a = constant(1,2,1,f64);
    bT = readData_Radiation(1);
    ATA = matmul(a,transpose(a));
    ATB = matmul(a,bT);
    tr_BTB = norm(bT);
    tr_BTB *= tr_BTB;
    num_points = 1;

    for(int i=2;i<N+1;i++){
        
        a(0,0) = i;
        bT = readData_Radiation(i);
        num_points += 1;

        if (num_points == 2){
            cur_loss=loss(ATB, ATA, bT, a, tr_BTB, true);
        }
        else{
            cur_loss=loss(ATB, ATA, bT, a, tr_BTB, false);
        }

        if(cur_loss > alpha or i == N){
            if(i == N){
                acc_loss += cur_loss;
                continue;
            }
            if(i < N){
                acc_loss += prev_loss;
                solu.push_back(i-1);
            }
            ATA = matmul(a, transpose(a));
            ATB = matmul(a, bT);
            tr_BTB = norm(bT);
            tr_BTB *= tr_BTB;
            cur_loss = 0;
            prev_loss = 0;
            num_points = 1;
        }else{
            prev_loss = cur_loss;
            
        }
    }

    for(auto iter = solu.begin() ; iter != solu.end() ; ++iter){      
        cout<<*iter<<" ";
    }
    cout<<endl;

    return acc_loss;
}

int intlog(double base, double x) {
    return (int)(log(x) / log(base));
}

int main(void){

    setDevice(0);
    int num = 0;
    double alpha = 0;
    double acc_loss = 0;
    double base = 5.0;

    alpha = pow(base, 21);
    cout <<"Therashold: " << alpha << endl;
    acc_loss = greedy_construct(alpha, num);
    cout <<"Loss: "<<acc_loss << endl;
    cout <<"Number of Segments: "<< num << endl;
    
    return 0;
}
