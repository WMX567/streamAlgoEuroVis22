#include "hip/hip_runtime.h"
#include <iostream>  
#include <math.h>
#include <arrayfire.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <fstream>
#include <algorithm> 
#include <chrono> 
#include <unistd.h>

using namespace af;
using namespace std;
using namespace std::chrono;

const int N = 48;
const int dim = 500*500*100;
double time_io = 0;
string dir = "/scratch/mw4355/Isabel/";
//********************************* In-Core ******************************************
auto data = new float[N][dim];

af::array readData_Radiation(int file_num){
    string num;
    FILE * pFile;

    file_num = file_num-1;
    if (file_num < 10){
        num = "000" + to_string(file_num);
    }else if(file_num < 100){
        num = "00" + to_string(file_num);
    }else{
        num= "0" + to_string(file_num);
    }
    string path = dir+"temperature"+num+".raw";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        if(data[i] > 23000 or data[i] < 70){
            data_[i] = 0;
        }else{
            data_[i] = data[i];
        }
    }
    fclose(pFile);
    af::array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

array readData_TeraShake(int file_num){
    FILE* pFile;
    string path = dir + "TeraShake" + to_string(file_num) + ".bin";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        data_[i] = data[i];
    }
    fclose(pFile);
    array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

void readData_Isabel(){
    FILE * pFile;
    string num;
    float* d = new float[dim];

    for(int i = 0; i < N; ++i){
        if (i+1 < 10)
            num = to_string(0)+to_string(i+1);
        else
            num= to_string(i+1);

        string path = dir+"TCf"+num+"converted.bin";
        TimerStart(0);
        pFile = fopen(path.c_str(),"rb");
        fread(d, dim*sizeof(float), 1, pFile);
        TimerStop(0);
        time_io +=  TimerGetRTime(0);

        for(int j = 0; j < dim; j++){
            data[i][j] = d[j];
            if(d[j] > 100 or d[j] < -100)
                data[i][j] = 100;
        }

        fclose(pFile);
    }
    delete [] d;
}

array getOne_Isabel(int i){
    double* data_ = new double[dim];
    for(int j = 0; j < dim; j++){
        data_[j] = data[i][j];
    }
    array oneStep(1, dim, data_);
    delete [] data_;
    return oneStep;
}

void readData_Vortex(){
    FILE * pFile;
    double time = 0;
    string num;
    float* d = new float[dim];

    for(int i = 0; i < N; ++i){
        if(i+1 == 65){
            num = to_string(64);
        }else{
            num = to_string(i+1);
        }
        string path = dir+"vorts"+num+"converted.data";
        TimerStart(0);
        pFile = fopen(path.c_str(),"rb");
        fread(d, dim*sizeof(float), 1, pFile);
        TimerStop(0);
        time_io +=  TimerGetRTime(0);

        for(int j = 0; j < dim; j++){
            data[i][j] = d[j];
        }

        fclose(pFile);
    }
    delete [] d;
}

array getOne_Vortex(int i){
    double* data_ = new double[dim];
    for(int j = 0; j < dim; j++){
        if(i+1 == 65){data_[j] = data[i-1][j];}
        else{data_[j] = data[i][j];}
    }
    array oneStep(1, dim, data_);
    delete [] data_;
    return oneStep;
}
//********************************** In-Core *****************************************

double loss(array& ATB, array& ATA, array& bT, 
    array& a, double& tr_BTB, bool two_near){
    int d = bT.dims(1);
    double temp;
    double* inv_CPU_1 = new double[2];
    double* inv_CPU_2 = new double[2];
    double* ATB_CPU_1 = new double[d];
    double* ATB_CPU_2 = new double[d];
    temp = norm(bT);
    tr_BTB += temp*temp;
    ATB += matmul(a, bT);
    ATA += matmul(a, transpose(a));
    if (two_near){return 0;}
    array ATA_inv = inverse(ATA);
    ATB(0, span).host(ATB_CPU_1);
    ATB(1, span).host(ATB_CPU_2);
    ATA_inv(0, span).host(inv_CPU_1);
    ATA_inv(1, span).host(inv_CPU_2);
    double tr;
    for(int i = 0; i < d; i++){
        if(i == 0){
            tr = ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
            tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
            continue;
        }
        tr += ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
        tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
    }
    delete [] ATB_CPU_1;
    delete [] ATB_CPU_2;
    delete [] inv_CPU_1;
    delete [] inv_CPU_2;
    return tr_BTB - tr;
}

void compute_E(double (&E)[N+1][N+1], string dir){
    array ATA, ATB, a, bT;
    double tr_BTB;
    //Compute E
    for(int i=1; i < N+1; i++){
        a = constant(1,2,1,f64);
        a(0,0) = i;
        //bT = getOne_Vortex(i,dir);
        bT = getOne_Isabel(i-1);
        ATA = matmul(a, transpose(a));
        ATB = matmul(a, bT);
        tr_BTB = norm(bT);
        tr_BTB *= tr_BTB;
        for(int j=i; j<N+1; j++){
            if (j == i){E[i][j] = 0; continue;}
            a(0,0) = j;
            bT = getOne_Isabel(j-1);
            if (j == i+1){E[i][j] = loss(ATB, ATA, bT, a, tr_BTB, true);}
            else{E[i][j] = loss(ATB, ATA, bT, a, tr_BTB, false);}
        }
    }
}

void dp_solu(double (&E)[N+1][N+1], double (&E_s)[N+1][N+1], int (&solu)[N+1][N+1]){
    for(int k=1; k<N+1; k++){
        for(int i=k;i<N+1; i++){
            if(k == 1){
                E_s[i][1] = E[1][i];
                solu[i][1] = i;
                continue;
            }
            for(int j=k; j < i+1; j++){
                if(j == k){ 
                    E_s[i][k] = E_s[j-1][k-1]+E[j][i];
                    solu[i][k] = j-1;
                }
                else{ 
                    if(E_s[i][k] > E_s[j-1][k-1]+E[j][i]){
                        E_s[i][k] = E_s[j-1][k-1]+E[j][i];
                        solu[i][k] = j-1;
                    }
                }
            }
        }
    }
}

int main(void){
    
    TimerStart(3);
    double E[N+1][N+1];
    double E_s[N+1][N+1];
    int solu[N+1][N+1];

    setDevice(0);
    readData_Isabel();

    TimerStart(1);
    compute_E(E,dir);
    TimerStop(1);
    cout << "Preprocess Time:" << TimerGetRTime(1) << endl;

    auto start = high_resolution_clock::now();
    dp_solu(E, E_s, solu);
    auto stop = high_resolution_clock::now();
    auto duration = duration_cast<microseconds>(stop - start); 
    cout << "DP Time:" << duration.count() << endl;
    TimerStop(3);
    cout << "Total Runtime: "<< TimerGetRTime(3) << endl;
    cout << "IO Time: " <<  time_io << endl;
    
    return 0;
}
