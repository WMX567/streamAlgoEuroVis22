#include "hip/hip_runtime.h"
#include <iostream>  
#include <math.h>
#include <arrayfire.h>
#include <stdio.h>
#include <stdlib.h>
#include "timer.h"
#include <fstream>
#include <unistd.h>
#include <limits>
#include <unordered_map>
#include <condition_variable>
#include <cmath>
#include <thread>
#include <mutex>
#include <vector>
using namespace af;
using namespace std;

const int N = 800;
const int n = 200;
const string dir = "/scratch/mw4355/Radiation/";
const int dim = 600*248*248;
double time_io = 0;

// Save result
ofstream savedFile("Radiation800_saved.txt");

//+++++++++++++++++++++++++ Reading Functions Area+++++++++++++++++++++++++++++
af::array readData_Isabel(int file_num){
    string num;
    FILE * pFile;
    if (file_num < 10){
        num = "0"+to_string(file_num);
    }else{
        num=to_string(file_num);
    }
    string path = dir+"TCf"+num+"converted.bin";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io += TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        if(data[i] > 100 or data[i] < -100){
            data_[i] = 100;
        }
        else{
            data_[i] = data[i];
        }
    }
    fclose(pFile);
    af::array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

af::array readData_Vortex(int file_num){
    string num;
    FILE * pFile;
    if(file_num == 65){
        num = to_string(64);
    }else if(file_num == 24){
        num = to_string(25);
    }else{
        num = to_string(file_num);
    }
    string path = dir+"vorts"+num+"converted.data";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        data_[i] = data[i];
    }
    fclose(pFile);
    af::array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

af::array readData_Radiation(int file_num){
    string num;
    FILE * pFile;
    file_num = file_num-1;
    if (file_num < 10){
        num = "000" + to_string(file_num);
    }else if(file_num < 100){
        num = "00" + to_string(file_num);
    }else{
        num= "0" + to_string(file_num);
    }
    string path = dir+"temperature"+num+".raw";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        if(data[i] > 23000 or data[i] < 70){
            data_[i] = 0;
        }else{
            data_[i] = data[i];
        }
    }
    fclose(pFile);
    af::array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}

af::array readData_TeraShake(int file_num){
    FILE* pFile;
    string path = dir + "TeraShake" + to_string(file_num) + ".bin";
    float* data = new float[dim];
    TimerStart(0);
    pFile = fopen(path.c_str(),"rb");
    fread(data, dim*sizeof(float), 1, pFile);
    TimerStop(0);
    time_io +=  TimerGetRTime(0);
    double* data_ = new double[dim];
    for(int i = 0; i < dim; i++){
        data_[i] = data[i];
    }
    fclose(pFile);
    af::array oneStep(1, dim, data_);
    delete[] data;
    delete[] data_;
    return oneStep;
}
//++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++++

//+++++++++++++++++++++++++ Algorithm ++++++++++++++++++++++++++++++++++++++++
double loss(af::array& ATB, af::array& ATA, af::array& bT, 
    af::array& a, double& tr_BTB, bool two_near){
    int d = bT.dims(1);
    double temp;
    double* inv_CPU_1 = new double[2];
    double* inv_CPU_2 = new double[2];
    double* ATB_CPU_1 = new double[d];
    double* ATB_CPU_2 = new double[d];
    temp = norm(bT);
    tr_BTB += temp*temp;
    ATB += matmul(a, bT);
    ATA += matmul(a, transpose(a));
    if (two_near){return 0;}
    af::array ATA_inv = inverse(ATA);
    ATB(0, span).host(ATB_CPU_1);
    ATB(1, span).host(ATB_CPU_2);
    ATA_inv(0, span).host(inv_CPU_1);
    ATA_inv(1, span).host(inv_CPU_2);
    double tr;
    for(int i = 0; i < d; i++){
        if(i == 0){
            tr = ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
            tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
            continue;
        }
        tr += ATB_CPU_1[i]*(ATB_CPU_1[i]*inv_CPU_1[0] + inv_CPU_2[0]*ATB_CPU_2[i]);
        tr += ATB_CPU_2[i]*(ATB_CPU_1[i]*inv_CPU_1[1] + inv_CPU_2[1]*ATB_CPU_2[i]);
    }
    delete [] ATB_CPU_1;
    delete [] ATB_CPU_2;
    delete [] inv_CPU_1;
    delete [] inv_CPU_2;
    return tr_BTB - tr;
}


//+++++++++++++++++++++++++ Helper Function ++++++++++++++++++++++++++++++++++++++++
//Parallel Computing
af::array shared_bT = readData_Radiation(1);
af::array shared_a = constant(1,2,1,f64);
int test_a = 1;
condition_variable cond;
mutex m_lock;
int counter = 0;
vector<thread> threads;
unordered_map<int, int> hash_;
double Err1 = 0;

int get_k(int i){
    if(i % n == 0){
        return n;
    }else if (i / n % 2 == 1){
        return (i % n) + (n+1)-2*(i % n);
    }else{
        return i % n;
    }
}

void keep_going_or_not(int prev_i){
    unique_lock<mutex> lk(m_lock);
    cond.wait(lk, [prev_i]{return test_a == prev_i+1;});
    lk.unlock();
    cond.notify_all();
}

void update_counter(){
    //Change Counter
    unique_lock<mutex> lk(m_lock);
    counter += 1;
    lk.unlock();
    cond.notify_all();
}

int intlog(double base, double x) {
    return (int)(log(x) / log(base));
}

void reading(){
    for(int i = 2; i < N+1; ++i){
        unique_lock<mutex> lk(m_lock);
        cond.wait(lk,[]{return counter == threads.size();});
        int k = get_k(i);
        shared_bT = readData_Radiation(k); //Read Data
        shared_a(0,0) = i;
        test_a = i;
        counter = 0;
        lk.unlock();
        cond.notify_all();
    }
}

void greedy_construct(double prev_loss, double alpha, 
int start, vector<int> solu){
    af::array ATA, ATB;
    int num_points = 0;
    int prev_i = start-1;
    double acc_loss = prev_loss;
    double cur_loss = 0;
    double tr_BTB = 0;
    for(int i=start;i<N+1;i++){
        keep_going_or_not(prev_i);
        prev_i += 1;
        if(i == start){
            prev_loss = 0;
            ATA = matmul(shared_a,transpose(shared_a));
            ATB = matmul(shared_a,shared_bT);
            tr_BTB = norm(shared_bT);
            tr_BTB *= tr_BTB;
            num_points = 1;
            update_counter(); //Update Counter
            continue;
        }
    
        num_points += 1;
        if (num_points == 2){
            cur_loss=loss(ATB, ATA, shared_bT, shared_a, tr_BTB, true);
        }
        else{
            cur_loss=loss(ATB, ATA, shared_bT, shared_a, tr_BTB, false);
        }
        if(cur_loss > alpha or i == N){
            if(i == N){
                acc_loss += cur_loss;
                continue;
            }
            if(i < N){
                acc_loss += prev_loss;
                solu.push_back(i-1);
            }
            ATA = matmul(shared_a, transpose(shared_a));
            ATB = matmul(shared_a, shared_bT);
            tr_BTB = norm(shared_bT);
            tr_BTB *= tr_BTB;
            cur_loss = 0;
            prev_loss = 0;
            num_points = 1;
        }else{
            prev_loss = cur_loss;
            
        }
        //Change Counter
        update_counter();
    }
    unique_lock<mutex> lk(m_lock);
    savedFile << alpha << ":" << solu.size()+1 << ":" << acc_loss << endl;
    lk.unlock();
    cond.notify_all();
}

void update_thread(double prev_loss, double alpha, 
int start, vector<int> solu){
    unique_lock<mutex> lk(m_lock);
    threads.push_back(thread(greedy_construct, prev_loss, alpha, start, solu));
    lk.unlock();
    cond.notify_all();
}

void compute_max(){
    af::array  ATA_1, ATB_1;
    double tr_BTB;
    int prev_i = 0;

    for(int i = 1; i < N+1; i++){
        keep_going_or_not(prev_i);
        prev_i += 1;
        if(i == 1){
            ATA_1 = matmul(shared_a,transpose(shared_a));
            ATB_1 = matmul(shared_a,shared_bT);
            tr_BTB = norm(shared_bT);
            tr_BTB *= tr_BTB;
            //Update Counter
            update_counter();
            continue;
        }

        if(i==2){
            Err1 = loss(ATB_1, ATA_1, shared_bT, shared_a, tr_BTB, true);
        }else{
            Err1 = loss(ATB_1, ATA_1, shared_bT, shared_a, tr_BTB, false);
        }

        update_counter();

    }

    unique_lock<mutex> lk(m_lock);
    savedFile << Err1 << ":"<< 1 <<":"<<Err1<<endl;
    lk.unlock();
    cond.notify_all();

}


int main(void){
    setDevice(0);
    TimerStart(4);
    double Err2 = 0;
    af::array ATA, ATB;
    double upper = 0;
    double lower = numeric_limits<double>::infinity();
    double base = 5.0;
    double t = 0.000001;
    double tr_BTB;
    int e_1 = 0;
    int e_2 = 0;
    int l_seg = 1;
    int prev_i = 0;
    vector<int> zero_solu;

    {
        unique_lock<mutex> lk(m_lock);
        threads.push_back(thread(reading));
        threads.push_back(thread(compute_max));
        lk.unlock();
        cond.notify_all();
    }
    
    for(int i = 1; i < N+1; i++){
        keep_going_or_not(prev_i);
        prev_i += 1;
        if(i == 1){
            //Data Reading
            ATA = matmul(shared_a,transpose(shared_a));
            ATB = matmul(shared_a,shared_bT);
            tr_BTB = norm(shared_bT);
            tr_BTB *= tr_BTB;
            //Update Counter
            update_counter();
            continue;
        }

        if(l_seg < 2){
            Err2 = loss(ATB, ATA, shared_bT, shared_a, tr_BTB, true);
            l_seg += 1;
        }else{
            Err2 = loss(ATB, ATA, shared_bT, shared_a, tr_BTB, false);
            l_seg += 1;
        }

        while(Err1 == upper && Err1 != 0){}

        if(Err1 > t){
            e_2 = intlog(base, Err1);
            if(upper == 0){
                e_1 = e_2;
            }else{
                e_1 = intlog(base, upper)+1;
                if(intlog(base, upper) == log(upper) / log(base)){
                    e_1 -= 1;
                }
            }
            for(int j = e_1; j <= e_2; j++){
                if(hash_.find(j) == hash_.end()){
                    hash_.insert(pair<int,int>(j,1));
                    vector<int> solu;
                    solu.push_back(i-1);
                    threads.push_back(thread(greedy_construct, upper, pow(base, j),i,solu));
                }
            }
            upper = Err1;
        }

       if(Err2 > t){
            zero_solu.push_back(i-1);
            ATA = matmul(shared_a,transpose(shared_a));
            ATB = matmul(shared_a,shared_bT);
            tr_BTB = norm(shared_bT);
            tr_BTB *= tr_BTB;
            l_seg = 1;
            if(lower > Err2){
                e_1 = intlog(base, Err2)+1;
                if(intlog(base, Err2) == log(Err2) / log(base)){
                    e_1 -= 1;
                }
                if(lower == numeric_limits<double>::infinity()){
                    e_2 = e_1;
                }else{
                    e_2 = intlog(base, lower);
                }
                for(int j= e_1; j <= e_2; j++){
                    if(hash_.find(j) == hash_.end()){
                        hash_.insert(pair<int,int>(j,1));
                        zero_solu[zero_solu.size()-1] = i;
                        threads.push_back(thread(greedy_construct, Err2, pow(base, j), i+1, zero_solu));
                        update_counter();
                        zero_solu[zero_solu.size()-1] = i-1;
                    }
                }
                lower = Err2;
            }
        }

        update_counter();
    }

    for(auto &th: threads){
        th.join();
    }

    TimerStop(4);
    cout << "Time: " << TimerGetRTime(4) << endl;
    cout << "IO Time: " << time_io << endl;
    unique_lock<mutex> lk(m_lock);
    savedFile << 0 << ":"<< zero_solu.size()+1 << ":"<<0<<endl;
    lk.unlock();
    cond.notify_all();
    savedFile.close();
    return 0;
    
}
